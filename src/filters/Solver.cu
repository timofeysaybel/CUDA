#include "hip/hip_runtime.h"
//
// Created by timofey
//

#include "../../include/filters/Solver.cuh"
#include "../../include/filters/Filters.h"
#include "../../include/io/Reader.h"
#include "../../include/io/Writer.h"

#include <iostream>
#include <cmath>

#define SAFE_CALL(CallInstruction) { \
    hipError_t cuerr = CallInstruction; \
    if(cuerr != hipSuccess) { \
         printf("CUDA error: %s at call \"" #CallInstruction "\"\n", hipGetErrorString(cuerr)); \
         throw "error in CUDA API function, aborting..."; \
    } \
}

#define SAFE_KERNEL_CALL(KernelCallInstruction){ \
    KernelCallInstruction; \
    hipError_t cuerr = hipGetLastError(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel launch: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel launch, aborting..."; \
    } \
    cuerr = hipDeviceSynchronize(); \
    if(cuerr != hipSuccess) { \
        printf("CUDA error in kernel execution: %s at kernel \"" #KernelCallInstruction "\"\n", hipGetErrorString(cuerr)); \
        throw "error in CUDA kernel execution, aborting..."; \
    } \
}

#define FRACTION_CEILING(numerator, denominator) ((numerator+denominator-1)/denominator)

const int BLOCK_SIZE = 16;
const int BLOCK3 = 14;
const int BLOCK5 = 12;

const int FILES_N = 100;

using namespace std;

//-----------------------------------------------------TASK 1-----------------------------------------------------------

__global__
void applyFilter(Pixel *image, Pixel *filtered, const double *kernel, int kernelCenter, int width, int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    double r = 0., g = 0., b = 0.;

    for (int i = -kernelCenter; i <= kernelCenter; i++)
    {
        for (int j = -kernelCenter; j <= kernelCenter; j++)
        {
            int xx = x + i;
            int yy = y + j;
            if (xx < 0)
                xx = 0;
            if (xx >= width)
                xx = width - 1;
            if (yy < 0)
                yy = 0;
            if (yy >= height)
                yy = height - 1;

            Pixel currentPixel = image[yy * width + xx];
            double currentKernelElement = kernel[(2 * kernelCenter + 1) * (j + kernelCenter) + i + kernelCenter];
            r += currentPixel.r * currentKernelElement;
            g += currentPixel.g * currentKernelElement;
            b += currentPixel.b * currentKernelElement;
        }
    }

    filtered[width * y + x].r = (unsigned char) round(r);
    filtered[width * y + x].g = (unsigned char) round(g);
    filtered[width * y + x].b = (unsigned char) round(b);
}

void Task1::Solver::solve(int filter, const std::string &inFilename, const std::string &outFilename)
{
    const string FORMAT = ".jpg";
    Image image = Reader::read(inFilename);
    Image filtered{};

    switch (filter)
    {
        case Filters::GAUSSIAN:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::EDGE_DETECTION:
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::SHARPEN:
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::ALL:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename + "gaussian" + FORMAT);
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename + "edge" + FORMAT);
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename + "sharpen" + FORMAT);

        default:
            break;
    }
}

Image Task1::Solver::solveGaussian(const Image &image)
{
    Pixel *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (Pixel *) malloc(image.width * image.height * sizeof(Pixel));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * sizeof(Pixel)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * sizeof(Pixel)));
    SAFE_CALL(hipMalloc(&dKernel, 25 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, image.data, image.width * image.height * sizeof(Pixel), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::gaussianKernel, 25 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(32, 32);
    dim3 blocks(image.width / threads.x + 1, image.height / threads.y + 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 2, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * sizeof(Pixel), hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Gaussian blur: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Task1::Solver::solveEdge(Image image)
{
    Pixel *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (Pixel *) malloc(image.width * image.height * sizeof(Pixel));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * sizeof(Pixel)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * sizeof(Pixel)));
    SAFE_CALL(hipMalloc(&dKernel, 9 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, image.data, image.width * image.height * sizeof(Pixel), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::edgeKernel, 9 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(32, 32);
    dim3 blocks(image.width / threads.x + 1, image.height / threads.y + 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 1, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * sizeof(Pixel), hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Edge detection: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Task1::Solver::solveSharpen(Image image)
{
    Pixel *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (Pixel *) malloc(image.width * image.height * sizeof(Pixel));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * sizeof(Pixel)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * sizeof(Pixel)));
    SAFE_CALL(hipMalloc(&dKernel, 9 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, image.data, image.width * image.height * sizeof(Pixel), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::sharpenKernel, 9 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(32, 32);
    dim3 blocks(image.width / threads.x + 1, image.height / threads.y + 1);
    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 1, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * sizeof(Pixel), hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Sharpen: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

//-------------------------------------------------ОПТИМИИЗАЦИЯ (TASK 2)------------------------------------------------
//-----------------------------------------1) РАЗВЕРТКА МАССИВА ИЗОБРАЖЕНИЯ---------------------------------------------

void Opt1::Solver::solve(int filter, const std::string &inFilename, const std::string &outFilename)
{
    const string FORMAT = ".jpg";
    Image image = Reader::read(inFilename);
    Image filtered{};

    switch (filter)
    {
        case Filters::GAUSSIAN:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::EDGE_DETECTION:
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::SHARPEN:
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::ALL:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename + "gaussian" + FORMAT);
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename + "edge" + FORMAT);
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename + "sharpen" + FORMAT);

        default:
            break;
    }
}

__global__
void applyFilter(unsigned char *image, unsigned char *filtered, const double *kernel, int kernelCenter, int width,
                 int height)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    double r = 0., g = 0., b = 0.;

    for (int i = -kernelCenter; i <= kernelCenter; i++)
    {
        for (int j = -kernelCenter; j <= kernelCenter; j++)
        {
            int xx = x + i;
            int yy = y + j;
            if (xx < 0)
                xx = 0;
            if (xx >= width)
                xx = width - 1;
            if (yy < 0)
                yy = 0;
            if (yy >= height)
                yy = height - 1;

            double currentKernelElement = kernel[(2 * kernelCenter + 1) * (j + kernelCenter) + i + kernelCenter];
            r += image[yy * width + xx] * currentKernelElement;
            g += image[width * height + yy * width + xx] * currentKernelElement;
            b += image[2 * width * height + yy * width + xx] * currentKernelElement;
        }
    }

    filtered[width * y + x] = (unsigned char) round(r);
    filtered[width * y + x + width * height] = (unsigned char) round(g);
    filtered[width * y + x + 2 * width * height] = (unsigned char) round(b);
}

Image Opt1::Solver::solveGaussian(const Image &image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dKernel, 25 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::gaussianKernel, 25 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    dim3 blocks(FRACTION_CEILING(image.width * 3, BLOCK_SIZE), FRACTION_CEILING(image.height, BLOCK_SIZE));

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 2, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Gaussian blur: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Opt1::Solver::solveEdge(Image image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dKernel, 9 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::edgeKernel, 9 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    dim3 blocks(FRACTION_CEILING(image.width * 3, BLOCK_SIZE), FRACTION_CEILING(image.height, BLOCK_SIZE));

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 1, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Edge detection: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Opt1::Solver::solveSharpen(Image image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dKernel, 9 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::sharpenKernel, 9 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    dim3 blocks(FRACTION_CEILING(image.width * 3, BLOCK_SIZE), FRACTION_CEILING(image.height, BLOCK_SIZE));

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 1, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Sharpen: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

//-----------------------------------------2) ИСПОЛЬЗОВАНИЕ РАЗДЕЛЯЕМОЙ ПАМЯТИ------------------------------------------

void Opt2::Solver::solve(int filter, const std::string &inFilename, const std::string &outFilename)
{
    const string FORMAT = ".jpg";
    Image image = Reader::read(inFilename);
    Image filtered{};

    switch (filter)
    {
        case Filters::GAUSSIAN:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::EDGE_DETECTION:
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::SHARPEN:
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::ALL:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename + "gaussian" + FORMAT);
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename + "edge" + FORMAT);
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename + "sharpen" + FORMAT);

        default:
            break;
    }
}

namespace opt2
{
    __global__
    void applyFilter(unsigned char *image, unsigned char *filtered, const double *kernel, int kernelCenter, int width,
                     int height)
    {
        __shared__ unsigned buf[BLOCK_SIZE * BLOCK_SIZE * 3];

        int x = (blockDim.x - 2 * kernelCenter) * blockIdx.x + threadIdx.x - 2 * kernelCenter;
        int y = (blockDim.y - 2 * kernelCenter) * blockIdx.y + threadIdx.y - 2 * kernelCenter;

        if (x < 0 || y < 0 || x >= width || y >= height)
            return;

        int idx = y * width + x + threadIdx.z * width * height;
        buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] = image[idx];
        __syncthreads();

        if (threadIdx.x <= kernelCenter - 1 || threadIdx.y <= kernelCenter - 1 ||
            threadIdx.x >= blockDim.x - kernelCenter || threadIdx.y >= blockDim.y - kernelCenter)
            return;

        double c = 0.;

        for (int i = -kernelCenter; i <= kernelCenter; i++)
        {
            for (int j = -kernelCenter; j <= kernelCenter; j++)
            {
                double currentKernelElement = kernel[(2 * kernelCenter + 1) * (j + kernelCenter) + i + kernelCenter];
                c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + i) * blockDim.x + threadIdx.x + j] *
                     currentKernelElement;
            }
        }

        filtered[threadIdx.z * width * height + width * y + x] = (unsigned char) round(c);
    }
}

Image Opt2::Solver::solveGaussian(const Image &image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dKernel, 25 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::gaussianKernel, 25 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(image.width , BLOCK5), FRACTION_CEILING(image.height, BLOCK5), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((opt2::applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 2, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Gaussian blur: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Opt2::Solver::solveEdge(Image image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dKernel, 9 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::edgeKernel, 9 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(image.width, BLOCK3), FRACTION_CEILING(image.height, BLOCK3), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((opt2::applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 1, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Edge detection: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Opt2::Solver::solveSharpen(Image image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    double *dKernel;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dKernel, 9 * sizeof(double)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(dKernel, Filters::sharpenKernel, 9 * sizeof(double), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(image.width, BLOCK3), FRACTION_CEILING(image.height, BLOCK3), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((opt2::applyFilter<<<blocks, threads>>>(dImage, dFiltered, dKernel, 1, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dKernel));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Sharpen: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

//-----------------------------------------3) ВЫБОР ОПТИМАЛЬНОГО БЛОКА--------------------------------------------------
//----------------------------------------------------USE Opt2----------------------------------------------------------

//--------------------------------------4) ОПТИМИЗАЦИЯ ПРИМЕНЕНИЯ ФИЛЬТРА-----------------------------------------------
//--------------------------------------------4.1) РАЗВЕРКТА ЦИКЛОВ-----------------------------------------------------
//---------------------------4.2) ИСПОЛЬЗОВАНИЕ ОТДЕЛЬНЫХ ФУНКЦИЙ ДЛЯ КАЖДОГО ФИЛЬТРА ----------------------------------

void Opt4::Solver::solve(int filter, const std::string &inFilename, const std::string &outFilename)
{
    const string FORMAT = ".jpg";
    Image image = Reader::read(inFilename);
    Image filtered{};

    switch (filter)
    {
        case Filters::GAUSSIAN:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::EDGE_DETECTION:
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::SHARPEN:
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename);
            break;

        case Filters::ALL:
            filtered = solveGaussian(image);
            Writer::write(filtered, outFilename + "gaussian" + FORMAT);
            filtered = solveEdge(image);
            Writer::write(filtered, outFilename + "edge" + FORMAT);
            filtered = solveSharpen(image);
            Writer::write(filtered, outFilename + "sharpen" + FORMAT);

        default:
            break;
    }
}

namespace opt4
{
    __device__
    const double gaussianKernel[] =
            {
                    1 / 273., 4 / 273., 7 / 273., 4 / 273., 1 / 273.,
                    4 / 273., 16 / 273., 26 / 273., 16 / 273., 4 / 273.,
                    7 / 273., 26 / 273., 41 / 273., 26 / 273., 7 / 273.,
                    4 / 273., 16 / 273., 26 / 273., 16 / 273., 4 / 273.,
                    1 / 273., 4 / 273., 7 / 273., 4 / 273., 1 / 273.,
            };

    __device__
    const double edgeKernel[] =
            {
                    0, -1, 0,
                    -1, 4, -1,
                    0, -1, 0,
            };

    __device__
    const double sharpenKernel[] =
            {
                    -1, -1, -1,
                    -1, 9, -1,
                    -1, -1, -1
            };
}

__global__
void gaussianFilter(unsigned char *image, unsigned char *filtered, int width, int height)
{
    __shared__ unsigned buf[BLOCK_SIZE * BLOCK_SIZE * 3];

    int x = (blockDim.x - 4) * blockIdx.x + threadIdx.x - 4;
    int y = (blockDim.y - 4) * blockIdx.y + threadIdx.y - 4;

    if (x < 0 || y < 0 || x >= width || y >= height)
        return;

    int idx = y * width + x + threadIdx.z * width * height;
    buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] = image[idx];
    __syncthreads();

    if (threadIdx.x <= 1 || threadIdx.y <= 1 ||
        threadIdx.x >= blockDim.x - 2 || threadIdx.y >= blockDim.y - 2)
        return;

    double c = 0.;

    using opt4::gaussianKernel;

    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 2) * blockDim.x + threadIdx.x - 2] *
            gaussianKernel[0];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x - 2] *
            gaussianKernel[1];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x - 2] *
            gaussianKernel[2];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x - 2] *
            gaussianKernel[3];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 2) * blockDim.x + threadIdx.x - 2] *
            gaussianKernel[4];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 2) * blockDim.x + threadIdx.x - 1] *
            gaussianKernel[5];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x - 1] *
            gaussianKernel[6];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x - 1] *
            gaussianKernel[7];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x - 1] *
            gaussianKernel[8];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 2) * blockDim.x + threadIdx.x - 1] *
            gaussianKernel[9];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 2) * blockDim.x + threadIdx.x] *
            gaussianKernel[10];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x] *
            gaussianKernel[11];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] *
            gaussianKernel[12];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x] *
            gaussianKernel[13];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 2) * blockDim.x + threadIdx.x] *
            gaussianKernel[14];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 2) * blockDim.x + threadIdx.x + 1] *
            gaussianKernel[15];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x + 1] *
            gaussianKernel[16];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x + 1] *
            gaussianKernel[17];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x + 1] *
            gaussianKernel[18];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 2) * blockDim.x + threadIdx.x + 1] *
            gaussianKernel[19];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 2) * blockDim.x + threadIdx.x + 2] *
            gaussianKernel[20];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x + 2] *
            gaussianKernel[21];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x + 2] *
            gaussianKernel[22];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x + 2] *
            gaussianKernel[23];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 2) * blockDim.x + threadIdx.x + 2] *
            gaussianKernel[24];

    filtered[threadIdx.z * width * height + width * y + x] = (unsigned char) round(c);
}

__global__
void edgeFilter(unsigned char *image, unsigned char *filtered, int width, int height)
{
    __shared__ unsigned buf[BLOCK_SIZE * BLOCK_SIZE * 3];

    int x = (blockDim.x - 2) * blockIdx.x + threadIdx.x - 2;
    int y = (blockDim.y - 2) * blockIdx.y + threadIdx.y - 2;

    if (x < 0 || y < 0 || x >= width || y >= height)
        return;

    int idx = y * width + x + threadIdx.z * width * height;
    buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] = image[idx];
    __syncthreads();

    if (threadIdx.x <= 0 || threadIdx.y <= 0 ||
        threadIdx.x >= blockDim.x - 1 || threadIdx.y >= blockDim.y - 1)
        return;

    double c = 0.;
    using opt4::edgeKernel;
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x - 1] *
         edgeKernel[1];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x] *
            edgeKernel[3];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] *
            edgeKernel[4];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x] *
            edgeKernel[5];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x + 1] *
            edgeKernel[7];

    filtered[threadIdx.z * width * height + width * y + x] = (unsigned char) round(c);
}

__global__
void sharpenFilter(unsigned char *image, unsigned char *filtered, int width, int height)
{
    __shared__ unsigned buf[BLOCK_SIZE * BLOCK_SIZE * 3];

    int x = (blockDim.x - 2) * blockIdx.x + threadIdx.x - 2;
    int y = (blockDim.y - 2) * blockIdx.y + threadIdx.y - 2;

    if (x < 0 || y < 0 || x >= width || y >= height)
        return;

    int idx = y * width + x + threadIdx.z * width * height;
    buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] = image[idx];
    __syncthreads();

    if (threadIdx.x <= 0 || threadIdx.y <= 0 ||
        threadIdx.x >= blockDim.x - 1 || threadIdx.y >= blockDim.y - 1)
        return;

    double c = 0.;
    using opt4::sharpenKernel;
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x - 1] *
         sharpenKernel[0];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x - 1] *
            sharpenKernel[1];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x - 1] *
            sharpenKernel[2];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x] *
            sharpenKernel[3];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x] *
            sharpenKernel[4];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x] *
            sharpenKernel[5];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y - 1) * blockDim.x + threadIdx.x + 1] *
            sharpenKernel[6];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + threadIdx.y * blockDim.x + threadIdx.x + 1] *
            sharpenKernel[7];
    c += buf[threadIdx.z * BLOCK_SIZE * BLOCK_SIZE + (threadIdx.y + 1) * blockDim.x + threadIdx.x + 1] *
            sharpenKernel[8];

    filtered[threadIdx.z * width * height + width * y + x] = (unsigned char) round(c);
}

Image Opt4::Solver::solveGaussian(const Image &image)
{
    unsigned char *dImage, *dFiltered, *filtered;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(image.width, BLOCK5), FRACTION_CEILING(image.height, BLOCK5), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((gaussianFilter<<<blocks, threads>>>(dImage, dFiltered, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Gaussian blur: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Opt4::Solver::solveEdge(Image image)
{
    unsigned char *dImage, *dFiltered, *filtered;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(image.width, BLOCK3), FRACTION_CEILING(image.height, BLOCK3), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((edgeFilter<<<blocks, threads>>>(dImage, dFiltered, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Edge detection: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

Image Opt4::Solver::solveSharpen(Image image)
{
    unsigned char *dImage, *dFiltered, *filtered;

    filtered = (unsigned char *) malloc(image.width * image.height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.width * image.height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.width * image.height * 3 * sizeof(unsigned char)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char *img = image.getData();

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.width * image.height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(image.width, BLOCK3), FRACTION_CEILING(image.height, BLOCK3), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((sharpenFilter<<<blocks, threads>>>(dImage, dFiltered, image.width, image.height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.width * image.height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    Image res(filtered, image.width, image.height, image.channels);

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Sharpen: " << endl;
    cout << "Time for " << image.height << "x" << image.width << " image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.height << "x" << image.width << " image without copying: " << tmp << endl;

    return res;
}

//--------------------------------------5) ОПТИМИЗАЦИЯ МАЛЕНЬКИХ ИЗОБРАЖЕНИЙ--------------------------------------------

void Opt5::Solver::solve(const std::vector<std::string> &inFilenames, std::vector<std::string> &outFilenames)
{
    const string FORMAT = ".jpg";
    int startFileIdx = 0;
    int amountOfFiles = inFilenames.size();
    int filesOffset = min(FILES_N, amountOfFiles);
    while (amountOfFiles > 0)
    {
        vector<Image> images;
        for (int i = startFileIdx; i < filesOffset; i++)
            images.emplace_back(Reader::read(inFilenames[i]));

        amountOfFiles-=filesOffset - startFileIdx;

        vector<Image> filtered;

        filtered = solveGaussian(images);
        for (int i = startFileIdx; i < filesOffset; i++)
            Writer::write(filtered[i], outFilenames[i] + "gaussian" + FORMAT);

        filtered = solveEdge(images);
        for (int i = startFileIdx; i < filesOffset; i++)
            Writer::write(filtered[i], outFilenames[i] + "gaussian" + FORMAT);

        filtered = solveSharpen(images);
        for (int i = startFileIdx; i < filesOffset; i++)
            Writer::write(filtered[i], outFilenames[i] + "gaussian" + FORMAT);

        startFileIdx = filesOffset;
        filesOffset += min(FILES_N, amountOfFiles);
    }
}

vector<Image> Opt5::Solver::solveGaussian(const std::vector<Image> &image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    int height = image[0].height, width = image[0].width;
    filtered = (unsigned char *) malloc(image.size() * width * height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.size() * width * height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.size() * width * height * 3 * sizeof(unsigned char)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char* img;
    img = (unsigned char *) malloc(image.size() * width * height * 3 * sizeof(unsigned char));
    for (int i = 0; i < image.size(); i++)
    {
        unsigned char *tmpImg = image[i].getData();
        for (int j = 0; j < image[i].width * image[i].height * 3; i++)
            img[i * image[i].width * image[i].height * 3 + j] = tmpImg[j];
    }

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.size() * width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(width * image.size(), BLOCK5), FRACTION_CEILING(height, BLOCK5), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((gaussianFilter<<<blocks, threads>>>(dImage, dFiltered, width, height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.size() * width * height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    vector<Image> result;
    for (int i = 0; i < image.size(); i++)
    {
        unsigned char* tmpFiltered = (unsigned char*)malloc(image[i].height * image[i].width * 3 * sizeof(unsigned char));
        for (int j = 0; j < image[i].height * image[i].width * 3; j++)
            tmpFiltered[j] = filtered[i * image[i].height * image[i].width * 3 + j];
        result.emplace_back(tmpFiltered, image[i].width, image[i].height, image[i].channels);
    }

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Gaussian blur: " << endl;
    cout << "Time for " << image.size() << " 300x300 image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.size() << " 300x300 image without copying: " << tmp << endl;

    return result;
}

vector<Image> Opt5::Solver::solveEdge(std::vector<Image> image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    int height = image[0].height, width = image[0].width;
    filtered = (unsigned char *) malloc(image.size() * width * height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.size() * width * height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.size() * width * height * 3 * sizeof(unsigned char)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char* img;
    img = (unsigned char *) malloc(image.size() * width * height * 3 * sizeof(unsigned char));
    for (int i = 0; i < image.size(); i++)
    {
        unsigned char *tmpImg = image[i].getData();
        for (int j = 0; j < image[i].width * image[i].height * 3; i++)
            img[i * image[i].width * image[i].height * 3 + j] = tmpImg[j];
    }

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.size() * width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(width * image.size(), BLOCK3), FRACTION_CEILING(height, BLOCK3), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((edgeFilter<<<blocks, threads>>>(dImage, dFiltered, width * image.size(), height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.size() * width * height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    vector<Image> result;
    for (int i = 0; i < image.size(); i++)
    {
        unsigned char* tmpFiltered = (unsigned char*)malloc(image[i].height * image[i].width * 3 * sizeof(unsigned char));
        for (int j = 0; j < image[i].height * image[i].width * 3; j++)
            tmpFiltered[j] = filtered[i * image[i].height * image[i].width * 3 + j];
        result.emplace_back(tmpFiltered, image[i].width, image[i].height, image[i].channels);
    }

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Edge detection: " << endl;
    cout << "Time for " << image.size() << " 300x300 image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.size() << " 300x300 image without copying: " << tmp << endl;

    return result;
}

vector<Image> Opt5::Solver::solveSharpen(std::vector<Image> image)
{
    unsigned char *dImage, *dFiltered, *filtered;
    int height = image[0].height, width = image[0].width;
    filtered = (unsigned char *) malloc(image.size() * width * height * 3 * sizeof(unsigned char));
    SAFE_CALL(hipMalloc(&dImage, image.size() * width * height * 3 * sizeof(unsigned char)));
    SAFE_CALL(hipMalloc(&dFiltered, image.size() * width * height * 3 * sizeof(unsigned char)));

    hipEvent_t start, stop, startCopy, stopCopy;
    unsigned char* img;
    img = (unsigned char *) malloc(image.size() * width * height * 3 * sizeof(unsigned char));
    for (int i = 0; i < image.size(); i++)
    {
        unsigned char *tmpImg = image[i].getData();
        for (int j = 0; j < image[i].width * image[i].height * 3; i++)
            img[i * image[i].width * image[i].height * 3 + j] = tmpImg[j];
    }

    SAFE_CALL(hipEventCreate(&start));
    SAFE_CALL(hipEventCreate(&stop));
    SAFE_CALL(hipEventCreate(&startCopy));
    SAFE_CALL(hipEventCreate(&stopCopy));

    SAFE_CALL(hipEventRecord(startCopy));

    SAFE_CALL(hipMemcpy(dImage, img, image.size() * width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    SAFE_CALL(hipDeviceSynchronize());

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 3);

    dim3 blocks(FRACTION_CEILING(width * image.size(), BLOCK3), FRACTION_CEILING(height, BLOCK3), 1);

    SAFE_CALL(hipEventRecord(start));

    SAFE_KERNEL_CALL((sharpenFilter<<<blocks, threads>>>(dImage, dFiltered, width * image.size(), height)));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stop));
    SAFE_CALL(hipEventSynchronize(stop));

    SAFE_CALL(hipMemcpy(filtered, dFiltered, image.size() * width * height * 3 * sizeof(unsigned char),
                         hipMemcpyDeviceToHost));
    SAFE_CALL(hipDeviceSynchronize());

    SAFE_CALL(hipEventRecord(stopCopy));
    SAFE_CALL(hipEventSynchronize(stopCopy));

    vector<Image> result;
    for (int i = 0; i < image.size(); i++)
    {
        unsigned char* tmpFiltered = (unsigned char*)malloc(image[i].height * image[i].width * 3 * sizeof(unsigned char));
        for (int j = 0; j < image[i].height * image[i].width * 3; j++)
            tmpFiltered[j] = filtered[i * image[i].height * image[i].width * 3 + j];
        result.emplace_back(tmpFiltered, image[i].width, image[i].height, image[i].channels);
    }

    free(filtered);
    SAFE_CALL(hipFree(dImage));
    SAFE_CALL(hipFree(dFiltered));

    float tmp = 0.;
    SAFE_CALL(hipEventElapsedTime(&tmp, startCopy, stopCopy));
    cout << "Sharpen: " << endl;
    cout << "Time for " << image.size() << " 300x300 image with copying: " << tmp << endl;
    SAFE_CALL(hipEventElapsedTime(&tmp, start, stop));
    cout << "Time for " << image.size() << " 300x300 image without copying: " << tmp << endl;

    return result;
}